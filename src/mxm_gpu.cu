/*
 * mxm_gpu.cu
 *  @author azamat, mmin
 *  @since  July 13, 2012
 */

#include <stdio.h>
#include <hip/hip_runtime.h>


#define KERNEL  1
#define TILE   16 //autotune-able

extern "C" {
  void mxm_std_gpu_(double* a, int* m, double* b, int* n, double* c, int* p);
  void local_grad3_gpu_(
    double* u1r, double* u1s, double* u1t,
    double* u2r, double* u2s, double* u2t,
    double* u3r, double* u3s, double* u3t,
    double* u1 , double* u2 , double* u3 ,
    double* dxm, double* dxtm, int* n, int* nelts, int* rank);
  void curl_gpu_(
    double* u1r, double* u1s, double* u1t,
    double* u2r, double* u2s, double* u2t,
    double* u3r, double* u3s, double* u3t,
    double* rxmn,double* sxmn,double* txmn,
    double* rymn,double* symn,double* tymn,
    double* rzmn,double* szmn,double* tzmn,
    double* w1,  double* w2,  double* w3, double* w3m, int* nxyz, int* nelts);
}

// basic curl kernel impl
__global__ void curl_vanilla(
    double* rxmn,double* rymn,double* rzmn,
    double* sxmn,double* symn,double* szmn,
    double* txmn,double* tymn,double* tzmn,
    double* u1r, double* u1s, double* u1t,
    double* u2r, double* u2s, double* u2t,
    double* u3r, double* u3s, double* u3t,
    double* w3m, const int nxyz, const int nelts,
    double* w1,  double* w2,  double* w3){
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  double w3mk;
  int k=0;
  for(int e=0; e<nelts; e++){
    k=e*nxyz+tid;
    w3mk=w3m[k];

    w1[k]= w3mk*u3r[k]*rymn[k]
         + w3mk*u3s[k]*symn[k]
         + w3mk*u3t[k]*tymn[k]
         - w3mk*u2r[k]*rzmn[k]
         - w3mk*u2s[k]*szmn[k]
         - w3mk*u2t[k]*tzmn[k];

    w2[k]= w3mk*u1r[k]*rzmn[k]
         + w3mk*u1s[k]*szmn[k]
         + w3mk*u1t[k]*tzmn[k]
         - w3mk*u3r[k]*rxmn[k]
         - w3mk*u3s[k]*sxmn[k]
         - w3mk*u3t[k]*txmn[k];

    w3[k]= w3mk*u2r[k]*rxmn[k]
         + w3mk*u2s[k]*sxmn[k]
         + w3mk*u2t[k]*txmn[k]
         - w3mk*u1r[k]*rymn[k]
         - w3mk*u1s[k]*symn[k]
         - w3mk*u1t[k]*tymn[k];
  }
}

// basic multi-mxm impl
__global__ void mxm_vanilla(double* a, const int m, double* b, const int n, double* c, const int p
                           ,const int nelts, const int ldims){
  const int row=blockIdx.y*blockDim.y+threadIdx.y;
  const int col=blockIdx.x*blockDim.x+threadIdx.x;
  if(row<m && col<p){ //eliminate out-of-bounds threads
    if(nelts==1){ //single mxm
      double s=0.0;
      for(int k=0; k<n; k++){
        s+=a[k*m+row]*b[col*n+k];
      }
      c[col*m+row]=s;
    }else{ //multiple mxm's
      int lda=(ldims&0x1)*m*n //if a's bit (0x1) is set, its leading dim is of size m*n 
        , ldb=((ldims&0x2)>>1)*n*p
        , ldc=((ldims&0x4)>>2)*m*p
        , ldai=((ldims&0x8)>>3)*m*n //for a's inner dimension
        , ldci=((ldims&0x8)>>3)*m*p;
      for(int e=0; e<nelts; e++){ // might need to launch 1 thread per element
        if(ldims<8){ //no inner iterations
          double s=0.0;
          for(int k=0; k<n; k++){
            s+=a[e*lda+k*m+row]*b[e*ldb+col*n+k];
          }
          c[e*ldc+col*m+row]=s;
        }else{
          for(int i=0; i<m; i++){
            double s=0.0;
            for(int k=0; k<n; k++){
              s+=a[e*lda+i*ldai+k*m+row]*b[e*ldb+col*n+k];
            }
            c[e*ldc+i*ldci+col*m+row]=s;
          }
        }
      }
    }
  }
}


// mxm with 1D arrays
__global__ void mxm_1d(double* a, const int m, double* b, const int n, double* c, const int p){
  const int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i<m){
    for(int k=0; k<p; k++){
      double s=0.0;
      for(int j=0; j<n; j++){
        s+=a[j*m+i]*b[k*n+j];
      }
      c[k*m+i]=s;
    }
  }
}


// mxm with 2D arrays
__global__ void mxm_shared(double* a, const int m, double* b, const int n, double* c, const int p){
  __shared__ double as[TILE][TILE];
  __shared__ double bs[TILE][TILE];
  int bx=blockIdx.x, by=blockIdx.y, tx=threadIdx.x, ty=threadIdx.y;
  const int row=by*TILE+ty;
  const int col=bx*TILE+tx;
  double s=0.0;
  for(int t=0;t<m/TILE;t++){
    as[ty][tx]=a[col*m+t*TILE+tx];
    bs[ty][tx]=b[col*n+t*TILE+ty];
    __syncthreads();
    for(int k=0; k<TILE; k++){
      s+=as[ty][k]*bs[k][tx];
    }
    __syncthreads();
    c[col*m+row]=s;
  }
}


// globally-visible basic mxm implementation for small matrices
void mxm_std_gpu_(double* a, int* m, double* b, int* n, double* c, int* p){
  /*device variables*/
  double *dev_a, *dev_b, *dev_c;
  int sizeofA=*m*(*n)*sizeof(double)
    , sizeofB=*n*(*p)*sizeof(double)
    , sizeofC=*m*(*p)*sizeof(double);
  /*malloc and memcopy data H2D*/
  hipMalloc(&dev_a,sizeofA);
  hipMalloc(&dev_b,sizeofB);
  hipMalloc(&dev_c,sizeofC);
  hipMemcpy(dev_a,a,sizeofA,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,sizeofB,hipMemcpyHostToDevice);
  /*thread dimensions*/
  dim3 dimBlock, dimGrid;
#if KERNEL==1
  dimBlock.x=TILE; dimGrid.x=(*p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(*m+dimBlock.y-1)/dimBlock.y;
  mxm_vanilla<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p,1,0);
#elif KERNEL==2
  dimBlock.x=TILE; dimGrid.x=(*m+dimBlock.x-1)/dimBlock.x;
  mxm_1d<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p);
#else
  dimBlock.x=TILE; dimGrid.x=(*p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(*m+dimBlock.y-1)/dimBlock.y;
  mxm_shared<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p);
#endif
  /*memcopy D2H*/
  hipMemcpy(c,dev_c,sizeofC,hipMemcpyDeviceToHost);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  hipDeviceSynchronize();
}


// sets up the aggregated mxm kernel launch
void mxm_gpu2(double* a, int as, int m
             ,double* b, int bs, int n
             ,double* c, int cs, int p
             ,int nelts, int mask, int dev){
  hipSetDevice(dev);
  /*device variables*/
  double *dev_a, *dev_b, *dev_c;
  int sizeofA=as*sizeof(double)
    , sizeofB=bs*sizeof(double)
    , sizeofC=cs*sizeof(double);
  /*malloc and memcopy H2D*/
  hipMalloc(&dev_a,sizeofA);
  hipMalloc(&dev_b,sizeofB);
  hipMalloc(&dev_c,sizeofC);
  hipMemcpy(dev_a,a,sizeofA,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,sizeofB,hipMemcpyHostToDevice);
  /*thread grid dimensions*/
  dim3 dimBlock, dimGrid;
  dimBlock.x=TILE; dimGrid.x=(p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(m+dimBlock.y-1)/dimBlock.y;
  mxm_vanilla<<<dimGrid,dimBlock>>>(dev_a,m, dev_b,n, dev_c,p, nelts,mask);
  /*memcopy D2H*/
  hipMemcpy(c,dev_c,sizeofC,hipMemcpyDeviceToHost);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  hipDeviceSynchronize();
}


/**
 * Performs aggregated mxm for all elements at once.
 *
 * foreach e in 0..nelts
 *   u@r_{NxN^2} = d_{NxN} * u@_{NxN^2}^{e} // here @ is either 1, 2 or 3
 *   foreach k in 0..N
 *     u@s_{NxN}^{k} = u@_{NxN}^{k,e} * dt_{NxN}
 *   u@t_{N^2xN} = u@_{N^2xN}^{e} * dt_{NxN}
 */
void local_grad3_gpu_(double* u1r, double* u1s, double* u1t,  
                      double* u2r, double* u2s, double* u2t,  
                      double* u3r, double* u3s, double* u3t,  
                      double* u1 , double* u2 , double* u3 ,  
                      double* d  , double* dt , int* n, int* nelts, int* rank){
  int n2=*n*(*n), n3=*n*n2, npts=n3*(*nelts);

  int devs = 0;
  hipGetDeviceCount(&devs);
  int devid = *rank%2;
  if (devs==1) {
    //       d_{NxN}   *  u*_{NxN^2} = u*r_{NxN^2}   foreach e
    mxm_gpu2(d,n2,*n,     u1,npts,*n,  u1r,npts,n2,  *nelts,6, 0);
    mxm_gpu2(d,n2,*n,     u2,npts,*n,  u2r,npts,n2,  *nelts,6, 0);
    mxm_gpu2(d,n2,*n,     u3,npts,*n,  u3r,npts,n2,  *nelts,6, 0);
  
    //       u*_{NxN}  *  dt_{NxN}  =  u*s_{NxN}     foreach e,k
    mxm_gpu2(u1,npts,*n,  dt,n2,*n,    u1s,npts,*n,  *nelts,13, 0);
    mxm_gpu2(u2,npts,*n,  dt,n2,*n,    u2s,npts,*n,  *nelts,13, 0);
    mxm_gpu2(u3,npts,*n,  dt,n2,*n,    u3s,npts,*n,  *nelts,13, 0);
  
    //       u*_{N^2xN} * dt_{NxN}  =  u*t_{N^2xN}   foreach e
    mxm_gpu2(u1,npts,n2,  dt,n2,*n,    u1t,npts,*n,  *nelts,5, 0);
    mxm_gpu2(u2,npts,n2,  dt,n2,*n,    u2t,npts,*n,  *nelts,5, 0);
    mxm_gpu2(u3,npts,n2,  dt,n2,*n,    u3t,npts,*n,  *nelts,5, 0);
  } else {
    // todo: fork threads or do async launches
    //       d_{NxN}   *  u*_{NxN^2} = u*r_{NxN^2}   foreach e
    mxm_gpu2(d,n2,*n,     u1,npts,*n,  u1r,npts,n2,  *nelts,6, devid);
    mxm_gpu2(d,n2,*n,     u2,npts,*n,  u2r,npts,n2,  *nelts,6, devid);
    mxm_gpu2(d,n2,*n,     u3,npts,*n,  u3r,npts,n2,  *nelts,6, devid);
  
    //       u*_{NxN}  *  dt_{NxN}  =  u*s_{NxN}     foreach e,k
    mxm_gpu2(u1,npts,*n,  dt,n2,*n,    u1s,npts,*n,  *nelts,13, devid);
    mxm_gpu2(u2,npts,*n,  dt,n2,*n,    u2s,npts,*n,  *nelts,13, devid);
    mxm_gpu2(u3,npts,*n,  dt,n2,*n,    u3s,npts,*n,  *nelts,13, devid);
  
    //       u*_{N^2xN} * dt_{NxN}  =  u*t_{N^2xN}   foreach e
    mxm_gpu2(u1,npts,n2,  dt,n2,*n,    u1t,npts,*n,  *nelts,5, devid);
    mxm_gpu2(u2,npts,n2,  dt,n2,*n,    u2t,npts,*n,  *nelts,5, devid);
    mxm_gpu2(u3,npts,n2,  dt,n2,*n,    u3t,npts,*n,  *nelts,5, devid);
  }
}

// Sets up the curl kernel
void curl_gpu_(double* u1r, double* u1s, double* u1t,
               double* u2r, double* u2s, double* u2t,
               double* u3r, double* u3s, double* u3t,
               double* rxmn,double* sxmn,double* txmn,
               double* rymn,double* symn,double* tymn,
               double* rzmn,double* szmn,double* tzmn,
               double* w1,  double* w2,  double* w3, double* w3m, int* nxyz, int* nelts){
  /*device variables*/
  double *dev_rxmn, *dev_rymn, *dev_rzmn
        ,*dev_sxmn, *dev_symn, *dev_szmn
        ,*dev_txmn, *dev_tymn, *dev_tzmn
        ,*dev_u1r, *dev_u1s, *dev_u1t
        ,*dev_u2r, *dev_u2s, *dev_u2t
        ,*dev_u3r, *dev_u3s, *dev_u3t
        ,*dev_w1, *dev_w2, *dev_w3, *dev_w3m;
  int nptsz=*nxyz*(*nelts)*sizeof(double);
  /*malloc and memcopy H2D*/
  hipMalloc(&dev_rxmn,nptsz);
  hipMalloc(&dev_rymn,nptsz);
  hipMalloc(&dev_rzmn,nptsz);
  hipMalloc(&dev_sxmn,nptsz);
  hipMalloc(&dev_symn,nptsz);
  hipMalloc(&dev_szmn,nptsz);
  hipMalloc(&dev_txmn,nptsz);
  hipMalloc(&dev_tymn,nptsz);
  hipMalloc(&dev_tzmn,nptsz);
  hipMalloc(&dev_u1r, nptsz);
  hipMalloc(&dev_u1s, nptsz);
  hipMalloc(&dev_u1t, nptsz);
  hipMalloc(&dev_u2r, nptsz);
  hipMalloc(&dev_u2s, nptsz);
  hipMalloc(&dev_u2t, nptsz);
  hipMalloc(&dev_u3r, nptsz);
  hipMalloc(&dev_u3s, nptsz);
  hipMalloc(&dev_u3t, nptsz);
  hipMalloc(&dev_w3m, nptsz);
  hipMalloc(&dev_w1,  nptsz);
  hipMalloc(&dev_w2,  nptsz);
  hipMalloc(&dev_w3,  nptsz);
  hipMemcpy(dev_rxmn,rxmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_rymn,rymn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_rzmn,rzmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_sxmn,sxmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_symn,symn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_szmn,szmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_txmn,txmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_tymn,tymn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_tzmn,tzmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u1r, u1r, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u1s, u1s, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u1t, u1t, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u2r, u2r, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u2s, u2s, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u2t, u2t, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u3r, u3r, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u3s, u3s, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u3t, u3t, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_w3m, w3m, nptsz,hipMemcpyHostToDevice);
  /*thread grid dimensions*/
  dim3 dimBlock, dimGrid;
  dimBlock.x=*nxyz; dimGrid.x=(15+dimBlock.x-1)/dimBlock.x;
  curl_vanilla<<<dimGrid,dimBlock>>>(
    dev_rxmn,dev_rymn,dev_rzmn,
    dev_sxmn,dev_symn,dev_szmn,
    dev_txmn,dev_tymn,dev_tzmn,
    dev_u1r,dev_u1s,dev_u1t,
    dev_u2r,dev_u2s,dev_u2t,
    dev_u3r,dev_u3s,dev_u3t,
    dev_w3m,*nxyz,*nelts,
    dev_w1, dev_w2, dev_w3
  );
  hipMemcpy(w1,dev_w1,nptsz,hipMemcpyDeviceToHost);
  hipMemcpy(w2,dev_w2,nptsz,hipMemcpyDeviceToHost);
  hipMemcpy(w3,dev_w3,nptsz,hipMemcpyDeviceToHost);
  hipFree(dev_rxmn);
  hipFree(dev_rymn);
  hipFree(dev_rzmn);
  hipFree(dev_sxmn);
  hipFree(dev_symn);
  hipFree(dev_szmn);
  hipFree(dev_txmn);
  hipFree(dev_tymn);
  hipFree(dev_tzmn);
  hipFree(dev_u1r);
  hipFree(dev_u1s);
  hipFree(dev_u1t);
  hipFree(dev_u2r);
  hipFree(dev_u2s);
  hipFree(dev_u2t);
  hipFree(dev_u3r);
  hipFree(dev_u3s);
  hipFree(dev_u3t);
  hipFree(dev_w3m);
  hipFree(dev_w1);
  hipFree(dev_w2);
  hipFree(dev_w3);
  hipDeviceSynchronize();
}

